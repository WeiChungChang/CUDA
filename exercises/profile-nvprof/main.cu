
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void scaleKernel(float *src, float *dst, float scale)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] * scale;
}

int main()
{

    float *a_dev;
    float *b_dev;
    float *a = new float[128*100];
    float *b = new float[128*100];
    hipSetDevice(0); 

    hipMalloc(&a_dev, sizeof(float)*128*100);
    hipMalloc(&b_dev, sizeof(float)*128*100);

    hipMemcpy (a_dev, a, sizeof(float)*1000, hipMemcpyHostToDevice);

    scaleKernel<<<100, 128>>>(a_dev, b_dev, 4.0f);
    scaleKernel<<<50, 128>>>(a_dev, b_dev, 4.0f);
    scaleKernel<<<50, 128>>>(a_dev+50, b_dev+50, 4.0f);
    scaleKernel<<<100, 128>>>(a_dev, b_dev, 4.0f);
    hipDeviceSynchronize();

    hipMemcpy (b, b_dev, sizeof(int)*1000, hipMemcpyDeviceToHost);
}