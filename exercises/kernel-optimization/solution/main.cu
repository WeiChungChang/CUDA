#include "hip/hip_runtime.h"
#include <iostream>
#include <png.h>
#include "util.h"
#include <limits>
#include <hip/hip_runtime_api.h>
#include <chrono>
#include <algorithm>

using namespace std;

__device__ __host__ float de(Vec3 pos) ;

__device__ __host__ Vec3 getNormal(Vec3 pos) 
{
	Vec3 xDir = Vec3(1,0,0);
	Vec3 yDir = Vec3(0,1,0);
	Vec3 zDir = Vec3(0,0,1);
	return Vec3::normalize(Vec3(de(pos+xDir)-de(pos-xDir),
	                            de(pos+yDir)-de(pos-yDir),
	                            de(pos+zDir)-de(pos-zDir)));
}


// distance "estimation" function 
// solution: includes a lot of math functions, compile entire program with -use_fast_math
__device__ __host__ float de(Vec3 pos) 
{
    float cutoff = 2;
    float power = 8;
	Vec3 z = pos;
	float dr = 1.0;
	float r = 0.0;
	for (int i = 0; i < 10 ; i++) 
    {
		r = Vec3::length(z);
		if (r>cutoff) break;
		
		float theta = acosf(z.z/r);
		float phi = atanf(z.y);
		dr =  powf( r, power-1.0f)*power*dr + 1.0f;
		
		float zr = powf( r,power);
		theta = theta*power;
		phi = phi*power;
		
		z = Vec3(zr) * Vec3(sinf(theta)*cosf(phi), sinf(phi)*sinf(theta), cosf(theta));
		z = z + pos;
	}
	return 0.5f*logf(r)*r/dr;
}

__global__ void zero(float *arr, int size)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size)
        return;
    arr[i] = 0;
}

// solution: kernel originally included some "accidental" double precision variables which caused the entire expression to be promoted to double precision math.
__global__ void computeUV(int height, int width, float *uvxArr, float *uvyArr)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width)
        return;
    if(y >= height)
        return;

    float widthFloat = width;
    float heightFloat = height;

    float uvx = (tanf(3.14159265f / 4.0f)) * (2.0f*x - widthFloat) /  widthFloat;
    float uvy = (tanf(3.14159265f / 4.0f)) * ( heightFloat /  widthFloat) * (2.0f*y- heightFloat) /  heightFloat;
    uvxArr[(y*width)+x] = uvx;
    uvyArr[(y*width)+x] = uvy;
}

// take a look at what operations are done in "de"
// kernel execution divergence, change launch parameters to instead be 8x8 block
__global__ void trace(int height, int width, float *uvxArr, float *uvyArr, float *distance, Vec3 lookDirection, Vec3 camUp, Vec3 camRight, Vec3 cameraLocation)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width)
        return;
    if(y >= height)
        return;

    float uvx = uvxArr[(y*width)+x];
    float uvy = uvyArr[(y*width)+x];
	Vec3 rayDirection = Vec3::normalize(lookDirection + Vec3(uvx) * camUp + Vec3(uvy) * camRight);
    float totalDistance = 0;
    bool hit = false;;
    for(int iter= 0; iter < 128; iter++)
    {
        Vec3 p = cameraLocation + Vec3(totalDistance) * rayDirection;
        
        float currentDist = de(p);
        totalDistance += currentDist;
        if (totalDistance > 10) {
    	    totalDistance = INFINITY;
    	    break;
        }
        if (currentDist < 0.00001f) {
    	    hit = true;
    	    break;
        }
    }
    distance[(y*width)+x] = totalDistance;
}

// x and y are inverted causing issues with memory accesses
__global__ void shade(int height, int width, float *uvxArr, float *uvyArr, float *distance,
                      Vec3 lookDirection, Vec3 camUp, Vec3 camRight, Vec3 backgroundColor, Vec3 cameraLocation,
                      float *rawR, float *rawG, float *rawB)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width)
        return;
    if(y >= height)
        return;
        
    float uvx = uvxArr[(y*width)+x];
    float uvy = uvyArr[(y*width)+x];
    float dist = distance[(y*width)+x];

    Vec3 ret = backgroundColor;
    if(dist != INFINITY)
    {
        dist-=0.0001f;
        Vec3 rayDirection = Vec3::normalize(lookDirection + Vec3(uvx) * camUp + Vec3(uvy) * camRight);

        Vec3 hitPoint = cameraLocation + Vec3(dist) * rayDirection;
        Vec3 normal = getNormal(hitPoint);

        float lamb = 0.6;
        float spec = 0.2;
        Vec3 objectColor = Vec3(0.8,0.2,0.8);

        Vec3 toLight = ( Vec3(2,2,1) - hitPoint);
        toLight = Vec3::normalize(toLight);

        Vec3 lambIn = Vec3(lamb) * fabsf(Vec3::dot(normal, toLight));

        Vec3 specIn = Vec3(spec) * powf(fabsf(Vec3::dot(normal, Vec3::normalize(toLight - hitPoint))), 1);
        ret = clamp(((lambIn  * clamp(objectColor)) + specIn));

    }
    rawR[(y*width)+x] = ret.x;
    rawG[(y*width)+x] = ret.y;
    rawB[(y*width)+x] = ret.z;

}


// again some execution divergence
__global__ void globalIllumination(int height, int width, float *uvxArr, float *uvyArr, float *distance, 
    Vec3 lookDirection, Vec3 camUp, Vec3 camRight, Vec3 backgroundColor, Vec3 cameraLocation, 
    float *rawR, float *rawG, float *rawB)
{
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width)
        return;
    if(y >= height)
        return;
    float dist = distance[(y*width)+x];


    if(dist != INFINITY)
    {
        float uvx = uvxArr[(y*width)+x];
        float uvy = uvyArr[(y*width)+x];
    
        float gi;
        Vec3 rayDirection = Vec3::normalize(lookDirection + Vec3(uvx) * camUp + Vec3(uvy) * camRight);
        Vec3 hitPoint = cameraLocation + Vec3(dist) * rayDirection;
        Vec3 normal = getNormal(hitPoint);
        
        float totalDistance = 0;
        for(int i = 0; i < 10; i++)
        {
            Vec3 p = hitPoint + Vec3(totalDistance) * normal;
            float currentDist = de(p);
            totalDistance += currentDist;
        }
        gi = normalize(0.001,0,totalDistance);
        if (gi > 1)
            gi = 1;
        gi = 1-gi;
        rawR[(y*width)+x] *= gi;
        rawG[(y*width)+x] *= gi;
        rawB[(y*width)+x] *= gi;

    }

}

// strided memory accesses, caches will save you with modern GPUs but consider using shared memory to get better memory access patterns
__global__ void downsample(int height, int width, int scale, float *rawR, float *rawG, float *rawB, float *imageR, float *imageG, float *imageB)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width)
        return;
    if(y >= height)
        return;    

    int inX = x*scale;
    int inY = y*scale;

    float outR = 0;
    float outG = 0;
    float outB = 0;
    for(int i = 0; i < scale; i++)
    {
        for(int j = 0; j < scale; j++)
        {
            outR += __ldg(&rawR[((inY+i)*width*scale)+(inX+j)]);
            outG += __ldg(&rawG[((inY+i)*width*scale)+(inX+j)]);
            outB += __ldg(&rawB[((inY+i)*width*scale)+(inX+j)]);
        }
    }
    imageR[(y*width)+x] = outR/(float)(scale*scale);
    imageG[(y*width)+x] = outG/(float)(scale*scale);
    imageB[(y*width)+x] = outB/(float)(scale*scale);
}


int main() {
    int width = 1024;
    int height = 768;

    int scale = 2;

    int heightScale = scale*height;
    int widthScale = scale*width;

    float *rawR;
    float *rawG;
    float *rawB;
    float *uvxArr;
    float *uvyArr;
    float *imageR;
    float *imageG;
    float *imageB;
    
    float *d_imageR;
    float *d_imageG;
    float *d_imageB;
    
    
    float *distance;


    hipMalloc(&rawR, (width*scale)*(height*scale) * sizeof(float));
    hipMalloc(&rawG, (width*scale)*(height*scale) * sizeof(float));
    hipMalloc(&rawB, (width*scale)*(height*scale) * sizeof(float));
    hipMalloc(&uvxArr, (width*scale)*(height*scale) * sizeof(float));
    hipMalloc(&uvyArr, (width*scale)*(height*scale) * sizeof(float));
    hipMalloc(&distance, (width*scale)*(height*scale) * sizeof(float));
    
    hipMalloc(&d_imageR, width*height * sizeof(float));
    hipMalloc(&d_imageG, width*height * sizeof(float));
    hipMalloc(&d_imageB, width*height * sizeof(float));

    hipMemset(rawB, 0, (width*scale)*(height*scale));
    hipMemset(rawG, 0, (width*scale)*(height*scale));
    hipMemset(rawB, 0, (width*scale)*(height*scale));
    hipMemset(uvxArr, 0, (width*scale)*(height*scale));
    hipMemset(uvyArr, 0, (width*scale)*(height*scale));
    hipMemset(distance, 0, (width*scale)*(height*scale));
    
    hipMemset(d_imageR, 0, (width)*(height));
    hipMemset(d_imageG, 0, (width)*(height));
    hipMemset(d_imageB, 0, (width)*(height));


    hipHostMalloc(&imageR, width*height * sizeof(float));
    hipHostMalloc(&imageG, width*height * sizeof(float));
    hipHostMalloc(&imageB, width*height * sizeof(float));
    

    float max = 0;
    float min = 0;

    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            imageR[((y)*width)+(x)] = 0;
            imageG[((y)*width)+(x)] = 0;
            imageB[((y)*width)+(x)] = 0;
        }
    }

    Vec3 cameraLocation = {1.2f,1.2f,1.2f};
    Vec3 focus = {0,0,0};
    Vec3 worldUp = Vec3(0,1,0);
    Vec3 lookDirection = Vec3::normalize(focus - cameraLocation);
    Vec3 camUp = Vec3::normalize(Vec3::cross(worldUp, lookDirection));
    Vec3 camRight = Vec3::normalize(Vec3::cross(lookDirection, camUp));


    // compute UV 

    dim3 threadsUv;
    dim3 blocksUv;

    threadsUv.x = 32;
    threadsUv.y = 32;

    blocksUv.x = 1+ (widthScale/threadsUv.x);
    blocksUv.y = 1+ (heightScale/threadsUv.y);

    getErrorCuda((computeUV<<<blocksUv, threadsUv>>>(heightScale, widthScale, uvxArr, uvyArr)));

    // trace

    dim3 threadsTrace;
    dim3 blocksTrace;

    threadsTrace.x = 8;
    threadsTrace.y = 8;

    blocksTrace.x = 1 + (widthScale/threadsTrace.x);
    blocksTrace.y = 1 + (heightScale/threadsTrace.y);

    getErrorCuda((trace<<<blocksTrace, threadsTrace>>>(heightScale, widthScale, uvxArr, uvyArr, distance, lookDirection, camUp, camRight, cameraLocation)));

    Vec3 backgroundColor = {0.3f};


    // shade
    dim3 threadsShade;
    dim3 blocksShade;

    threadsShade.x = 32;
    threadsShade.y = 32;

    blocksShade.x = 1 + (widthScale/threadsShade.x);
    blocksShade.y = 1 + (heightScale/threadsShade.y);

    getErrorCuda((shade<<<blocksShade, threadsShade>>>(heightScale, widthScale, uvxArr, uvyArr, distance,
    lookDirection, camUp, camRight, backgroundColor, cameraLocation, rawR, rawG, rawB)));

    
    // global illumination

    dim3 threadsGi;
    dim3 blocksGi;

    threadsGi.x = 8;
    threadsGi.y = 8;

    blocksGi.x = 1+ (widthScale/threadsGi.x);
    blocksGi.y = 1+ (heightScale/threadsGi.y);

    getErrorCuda((globalIllumination<<<blocksGi, threadsGi>>>(heightScale, widthScale, uvxArr, uvyArr, distance, 
    lookDirection, camUp, camRight, backgroundColor, cameraLocation, rawR, rawG, rawB)));


    // downsample

    dim3 threadsDs;
    dim3 blocksDs;

    threadsDs.x = 16;
    threadsDs.y = 16;

    blocksDs.x = 1+ (width/threadsDs.x);
    blocksDs.y = 1+ (height/threadsDs.y);
    getErrorCuda((downsample<<<blocksDs, threadsDs>>>(height, width, scale, rawR, rawG, rawB, d_imageR, d_imageG, d_imageB)));

    hipDeviceSynchronize();

    hipMemcpy(imageR, d_imageR, sizeof(float)*height*width, hipMemcpyDefault);
    hipMemcpy(imageG, d_imageG, sizeof(float)*height*width, hipMemcpyDefault);
    hipMemcpy(imageB, d_imageB, sizeof(float)*height*width, hipMemcpyDefault);

    std::cout << "writing image" << std::endl;
    writeImageRGB("test.png", width,height, imageR,imageG,imageB, "output");

    return 0;
}